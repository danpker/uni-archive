/* Daniel Parker
 * University of Reading
 * 215 Parallel algorithms for Bioinformatics
 *
 * random.cu - generate some random strings for testing
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

void generate(char string[], int len) {

	int i;

	for (i = 0; i < len; i++) {
		string[i] = 'a' + (rand() % 4 + 1);
	}
}

